#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <sys/time.h>
#include <time.h>
#include <windows.h>
#include <random>
#include "dev.h"

inline void __cudaErrorCheck(hipError_t cudaStatus)
{
	cudaStatus = hipGetLastError();
    
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
}

int main(int argc, char *argv[])
{
	int i, j, k, l, t;
	int size = 0;
	float sum;
	FILE *fp = NULL, *record = NULL;
	int dim_len[DIM] = {DIM0_LEN, DIM1_LEN, DIM2_LEN/*, DIM3_LEN, DIM4_LEN*/};
	float *U, *V, *W;
	Ratings *r;
	float *r0; // the divergence between r and the approximation of r
	float temp;
//	struct timeval tv1, tv2;
	DWORD t1, t2;
	int temp1, temp2, temp3, temp4;
	float e;
	float grad0, grad1;
	int num_u[DIM0_LEN], num_v[DIM1_LEN], num_w[DIM2_LEN];
	int index_u[DIM0_LEN + 1], index_v[DIM1_LEN + 1];
	int in, out;
	int in_iter = NONZEROS_NUM, out_iter = 500;
//	int random[NONZEROS_NUM];

	Ratings *dev_r;
	float *dev_r0;
	float *dev_U, *dev_V, *dev_W;
	float *dev_up_U, *dev_up_V, *dev_up_W;
	float *dev_down_U, *dev_down_V, *dev_down_W;

	int loop_counts = 0;
	float pre_rmse = 10000000000.0, curr_rmse = 0.0;
//	float alpha = 0.000001;
	float alpha = 0.001;
	float lamda = 0.05;
	
	std::default_random_engine generator, generator1;
	std::uniform_int_distribution<int> distribution(0, NONZEROS_NUM - 1);
	std::uniform_real_distribution<float> distribution1(0.5, 1.5);

	hipError_t cudaStatus = hipGetLastError();

	if((fp = fopen("./file6", "r")) == NULL)
//	if((fp = fopen("ratings10m.dat", "r")) == NULL)
	{
		printf("cannot open this file!\n");
		exit(0);
	}
        if((record = fopen("record", "w")) == NULL)
        {
                printf("cannot open this file!\n");
                exit(0);
        }

	U = (float *)calloc(DIM0_LEN * ATTR, sizeof(float));
	V = (float *)calloc(DIM1_LEN * ATTR, sizeof(float));
	W = (float *)calloc(DIM2_LEN * ATTR, sizeof(float));
	r = (Ratings *)calloc(NONZEROS_NUM, sizeof(Ratings));
	r0 = (float *)calloc(NONZEROS_NUM, sizeof(float));
	
	printf("1\n");

	hipMalloc((void **)&dev_r, NONZEROS_NUM * sizeof(Ratings));
	hipMalloc((void **)&dev_r0, NONZEROS_NUM * sizeof(float));
	hipMalloc((void **)&dev_U, DIM0_LEN * ATTR * sizeof(float));
	hipMalloc((void **)&dev_V, DIM1_LEN * ATTR * sizeof(float));
	hipMalloc((void **)&dev_W, DIM2_LEN * ATTR * sizeof(float));
	hipMalloc((void **)&dev_up_U, DIM0_LEN * ATTR * sizeof(float));
	hipMalloc((void **)&dev_up_V, DIM1_LEN * ATTR * sizeof(float));
	hipMalloc((void **)&dev_up_W, DIM2_LEN * ATTR * sizeof(float));
	hipMalloc((void **)&dev_down_U, DIM0_LEN * ATTR * sizeof(float));
	hipMalloc((void **)&dev_down_V, DIM1_LEN * ATTR * sizeof(float));
	hipMalloc((void **)&dev_down_W, DIM2_LEN * ATTR * sizeof(float));
	
	__cudaErrorCheck(cudaStatus);
	
	printf("2\n");

	while((fscanf(fp, "%d %d %f %d %d %d %d", &r[size].row, &r[size].col, &r[size].rating, &temp1, &r[size].ctx, &temp2, &temp3)) != EOF) size ++;

//	printf("%d\n", size);

	for(i = 0; i < DIM0_LEN * ATTR; i++) U[i] = distribution1(generator1);
	for(i = 0; i < DIM1_LEN * ATTR; i++) V[i] = distribution1(generator1);
	for(i = 0; i < DIM2_LEN * ATTR; i++) W[i] = distribution1(generator1);

	for(i = 0; i < NONZEROS_NUM; i++)
	{
		r0[i] = 0;
		for(k = 0; k < ATTR; k++) r0[i] += U[r[i].row * ATTR + k] * V[r[i].col * ATTR + k] * W[r[i].ctx * ATTR + k];
	}
	for(i = 0; i < NONZEROS_NUM; i++) r0[i] = r[i].rating - r0[i];

	curr_rmse = 0.0;
	for(i = 0; i < NONZEROS_NUM; i++) curr_rmse += r0[i] / NONZEROS_NUM * r0[i];
	curr_rmse = sqrt(curr_rmse);
	fprintf(record, "%.6f\n", curr_rmse);
	
	hipMemcpy(dev_r, r, NONZEROS_NUM * sizeof(Ratings), hipMemcpyHostToDevice);
	hipMemcpy(dev_r0, r0, NONZEROS_NUM * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_U, U, DIM0_LEN * ATTR * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_V, V, DIM1_LEN * ATTR * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_W, W, DIM2_LEN * ATTR * sizeof(float), hipMemcpyHostToDevice);
	
	__cudaErrorCheck(cudaStatus);

	printf("3\n");

//	gettimeofday(&tv1,NULL);
	t1 = GetTickCount();

//	INIT_AVG_GRAD<<<9700, 1024>>>(dev_avg_grad_U, dev_avg_grad_V, dev_avg_grad_W);
	
	for(out = 0; out < out_iter; out++)
	{

		//<<<1700, 192>>>, <<<6000, 192>>>
		
//		INIT_AVG_GRAD<<<9700, 1024>>>(dev_avg_grad_U, dev_avg_grad_V, dev_avg_grad_W);
		// block_num * thread_num > DIM0_LEN * ATTR
		
//		COMPUTE_AVG_GRAD_ATOMIC<<<8000, 1024>>>(dev_r, dev_r0, dev_grad_U, dev_grad_V, dev_grad_W, dev_avg_grad_U, dev_avg_grad_V, dev_avg_grad_W, dev_num_u, dev_num_v, dev_num_w, dev_random);
		// block_num * thread_num > NONZEROS_NUM
		
//		SVRG_ATOMIC<<<8000, 1024>>>(dev_r, dev_r0, dev_U, dev_V, dev_W, dev_grad_U, dev_grad_V, dev_grad_W, dev_avg_grad_U, dev_avg_grad_V, dev_avg_grad_W, dev_random);
		
//		INIT_GRAD<<<9700, 1024>>>(dev_U, dev_V, dev_W, dev_grad_U, dev_grad_V, dev_grad_W);

		INITIALIZE_UP_DOWN<<<9700, 1024>>>(dev_up_U, dev_up_V, dev_up_W, dev_down_U, dev_down_V, dev_down_W);

		COMPUTE_DIM0<<<8000, 1024>>>(dev_r, dev_U, dev_V, dev_W, dev_up_U, dev_down_U);

		CUSNTF_DIM0<<<9700, 1024>>>(dev_U, dev_up_U, dev_down_U);
		
		COMPUTE_DIM1<<<8000, 1024>>>(dev_r, dev_U, dev_V, dev_W, dev_up_V, dev_down_V);
		
		CUSNTF_DIM1<<<1100, 1024>>>(dev_V, dev_up_V, dev_down_V);
		
		COMPUTE_DIM2<<<8000, 1024>>>(dev_r, dev_U, dev_V, dev_W, dev_up_W, dev_down_W);
		
		CUSNTF_DIM2<<<10, 1024>>>(dev_W, dev_up_W, dev_down_W);
		
		COMPUTE_R0<<<8000, 1024>>>(dev_r, dev_r0, dev_U, dev_V, dev_W);
		
		__cudaErrorCheck(cudaStatus);

		hipMemcpy(r0, dev_r0, NONZEROS_NUM * sizeof(float), hipMemcpyDeviceToHost);
		
		__cudaErrorCheck(cudaStatus);

		curr_rmse = 0.0;
		sum = 0;
		for(i = 0; i < NONZEROS_NUM; i++) 
		{
			temp = r0[i] / NONZEROS_NUM * r0[i];
			curr_rmse += temp;
/*		hipMemcpy(avg_grad_W, dev_avg_grad_W, DIM2_LEN * ATTR * sizeof(float), hipMemcpyDeviceToHost);
		for(i = 0; i < DIM2_LEN; i++) fprintf(record, "%d %.4f\n", i, avg_grad_W[i * ATTR]);
		fprintf(record, "\n\n");*/
/*			if( i > 5 && i < 10)
			{
				k = i;
				hipMemcpy(&U[r[k].row * ATTR], &dev_U[r[k].row * ATTR], ATTR * sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(&V[r[k].col * ATTR], &dev_V[r[k].col * ATTR], ATTR * sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(&W[r[k].ctx * ATTR], &dev_W[r[k].ctx * ATTR], ATTR * sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(&avg_grad_U[r[k].row * ATTR], &dev_avg_grad_U[r[k].row * ATTR], ATTR * sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(&avg_grad_V[r[k].col * ATTR], &dev_avg_grad_V[r[k].col * ATTR], ATTR * sizeof(float), hipMemcpyDeviceToHost);
				hipMemcpy(&avg_grad_W[r[k].ctx * ATTR], &dev_avg_grad_W[r[k].ctx * ATTR], ATTR * sizeof(float), hipMemcpyDeviceToHost);
				__cudaErrorCheck(cudaStatus);
				printf("***************\n");
				printf("%d  %.6f\n", i, sum);
				printf("%d  %d  %d\n", num_u[r[k].row], num_v[r[k].col], num_w[r[k].ctx]);
				for(j = 0; j < ATTR; j++) printf("%.6f ", U[r[k].row * ATTR + j]);
				printf("\n");
				for(j = 0; j < ATTR; j++) printf("%.6f ", V[r[k].col * ATTR + j]);
				printf("\n");
				for(j = 0; j < ATTR; j++) printf("%.6f ", W[r[k].ctx * ATTR + j]);
				printf("\n");
				for(j = 0; j < ATTR; j++) printf("%.6f ", avg_grad_U[r[k].row * ATTR + j]);
				printf("\n");
				for(j = 0; j < ATTR; j++) printf("%.6f ", avg_grad_V[r[k].col * ATTR + j]);
				printf("\n");
				for(j = 0; j < ATTR; j++) printf("%.6f ", avg_grad_W[r[k].ctx * ATTR + j]);
				printf("\n");
				printf("***************\n");
			}*/
		}
		curr_rmse = sqrt(curr_rmse);
		fprintf(record, "%.6f\n", curr_rmse);
//		printf("%d  %.6f\n", k, sum);

		loop_counts ++;
		if(/*curr_rmse >= pre_rmse ||*/ loop_counts >= 1000) break;
		pre_rmse = curr_rmse;
	}
//	gettimeofday(&tv2,NULL);
	t2 = GetTickCount();

//	fprintf(record, "average time per loop:%.4fms\n",((float)(tv2.tv_sec-tv1.tv_sec)*1000+(float)(tv2.tv_usec-tv1.tv_usec)/1000)/loop_counts);
	fprintf(record, "average time per loop:%.4fms\n",(float)(t2 - t1)/loop_counts);

	hipFree(dev_r);
	hipFree(dev_r0);
	hipFree(dev_U);
	hipFree(dev_V);
	hipFree(dev_W);
	hipFree(dev_up_U);
	hipFree(dev_up_V);
	hipFree(dev_up_W);
	hipFree(dev_down_U);
	hipFree(dev_down_V);
	hipFree(dev_down_W);

	free(U);
	free(V);
	free(W);
	free(r);
	free(r0);
	fclose(fp);
	fclose(record);
	return 0;
}
